// reading a text file
#include <iostream>
#include <fstream>
#include <string>

#include <hip/hip_runtime.h>

using namespace std;

__global__ void vector_simple_change(double *out, double *a, int n) {
    int index = threadIdx.x;
    int stride = blockDim.x;

    for(int i = index; i < n; i += stride){
        out[i] = a[i] + 1.0;
    }
}

int main () {
    int num_lines = 1000;
    double darray[num_lines];
    double out[num_lines]; // For storing final answer
    double value;

    int line_num = 0;
    string line;
    ifstream myfile ("data.txt");
    if (myfile.is_open())
    {
        while ( getline (myfile,line) )
        {
          //cout << line << '\n';
          printf("WORKING ON CURRENT LINE: %s\n", line.c_str());
          value = stod(line); // Convert to double
          printf("Setting darray[%d] to %f\n", line_num, value);
          darray[line_num] = value;
          line_num++;
        }
        myfile.close();
    } else
    {
        cout << "Unable to open file";
    }

    for (int array_index = 0; array_index < num_lines; array_index++) {
        printf("darray[%d] = %.17f\n", array_index, darray[array_index]);
    }

    // Allocate device memory on GPU
    double *device_darray;
    double *device_out;
    hipMalloc((void**)&device_darray, sizeof(double) * num_lines);
    hipMalloc((void**)&device_out, sizeof(double) * num_lines);

    hipMemcpy(device_darray, darray, sizeof(double) * num_lines, hipMemcpyHostToDevice);

    // Executing kernel
    vector_simple_change<<<1,256>>>(device_out, device_darray, num_lines);

    hipMemcpy(out, device_out, sizeof(double) * num_lines, hipMemcpyDeviceToHost);

    for (int array_index = 0; array_index < num_lines; array_index++) {
        printf("out[%d] = %.17f\n", array_index, out[array_index]);
    }

    hipFree(device_darray);
    hipFree(device_out);

    return 0;
}
